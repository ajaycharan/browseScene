#include "hip/hip_runtime.h"
#include<thrust/random.h>
#include<thrust/transform.h>
#include<thrust/device_vector.h>
#include<thrust/transform.h>
#include "add_ccd_noise.h"

/// Perlin noise: https://github.com/pabennett/glblox/blob/master/lib/perlin.cpp


__host__ __device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

struct ccd_camera_noise
{
    const float sigma_s_red;
    const float sigma_s_green;
    const float sigma_s_blue;

    const float sigma_c_red;
    const float sigma_c_green;
    const float sigma_c_blue;

    const float scale;

    ccd_camera_noise(float _sigma_s_red,
                    float _sigma_s_green,
                    float _sigma_s_blue,
                    float _sigma_c_red,
                    float _sigma_c_green,
                    float _sigma_c_blue,
                    float _scale) : sigma_s_red(_sigma_s_red),
       sigma_s_green(_sigma_s_green),
       sigma_s_blue(_sigma_s_blue),
       sigma_c_red(_sigma_c_red),
       sigma_c_green(_sigma_c_green),
       sigma_c_blue(_sigma_c_blue),
       scale(_scale)
       {}

  __host__ __device__  float4 operator()(const float4& val, const unsigned int& thread_id )
  {

      float4 noisy_pix;

      clock_t start_time = clock();

      unsigned int seed = hash(thread_id) + start_time;

      thrust::minstd_rand rng(seed);

      noisy_pix.x = val.x/scale;
      noisy_pix.y = val.y/scale;
      noisy_pix.z = val.z/scale;

      thrust::random::experimental::normal_distribution<float> red_pnoise  (0.0f,sqrt(val.x)*sigma_s_red  );
      thrust::random::experimental::normal_distribution<float> green_pnoise(0.0f,sqrt(val.y)*sigma_s_green);
      thrust::random::experimental::normal_distribution<float> blue_pnoise (0.0f,sqrt(val.z)*sigma_s_blue );

      thrust::random::experimental::normal_distribution<float> red_cnoise   (0.0f,sigma_c_red  );
      thrust::random::experimental::normal_distribution<float> green_cnoise (0.0f,sigma_c_green);
      thrust::random::experimental::normal_distribution<float> blue_cnoise  (0.0f,sigma_c_blue );

      noisy_pix.x = noisy_pix.x  + red_pnoise(rng)   + red_cnoise(rng);
      noisy_pix.y = noisy_pix.y  + green_pnoise(rng) + green_cnoise(rng);
      noisy_pix.z = noisy_pix.z  + blue_pnoise(rng)  + blue_cnoise(rng);

      noisy_pix.w = 1.0f;

      return noisy_pix;
  }
};


void launch_add_camera_noise(float4* img_array,
                             float4* noisy_image,
                             float4 sigma_s,
                             float4 sigma_c,
                             const unsigned int stridef4,
                             const unsigned int height,
                             float scale)
{
    thrust::device_ptr<float4>img_src(img_array);

    thrust::device_ptr<float4>img_dest(noisy_image);

    thrust::transform(img_src,img_src + stridef4*height, thrust::make_counting_iterator(0), img_dest,
                                                                  ccd_camera_noise(sigma_s.x,
                                                                                   sigma_s.y,
                                                                                   sigma_s.z,
                                                                                   sigma_c.x,
                                                                                   sigma_c.y,
                                                                                   sigma_c.z,
                                                                                   scale)
                                                                                   );
}

//__global__ void cu_GaussianConvolution(unsigned char *d_src,
//                                       unsigned char *d_dest,
//                                       float *d_kernel,
//                                       const unsigned int stridechar,
//                                       const unsigned int width,
//                                       const unsigned int height,
//                                       const int ksize)
//{

//    const unsigned int x = (blockIdx.x*blockDim.x + threadIdx.x);
//    const unsigned int y = (blockIdx.y*blockDim.y + threadIdx.y);

//    float d_val = 0;

//    for(int ix = -ksize/2; ix < ksize; ix++)
//    {
//        for(int iy = -ksize/2; iy < ksize; iy++)
//        {
//            if( x+ix>=0 && x+ix <width && y+iy>=0 && y+iy<height)
//            {
//                d_val += (float)d_src[(x+ix)+(y+iy)*stridechar]*d_kernel[ix+iy*ksize];
//            }
//        }
//    }

//    d_dest[x+y*stridechar] = (unsigned char)d_val;

//}

//void ApplyGaussianFilter(unsigned char* src,
//                         unsigned char* dest,
//                         const unsigned int stridechar,
//                         const unsigned int width,
//                         const unsigned int height,
//                         float sigma,
//                         int ksize,
//                         const dim3 block,
//                         const dim3 grid)
//{


//    float* d_kernel = new float[ksize*ksize];

//    float sigma_sqr = sigma*sigma;

//    float sum=0;

//    for(int ix = -ksize/2 ; ix< ksize/2 ; ix++)
//    {
//        for(int iy = -ksize/2 ; iy< ksize/2 ; iy++)
//        {
//            d_kernel[ix+iy*ksize] = expf(-(ix*ix+iy*iy)/(2*sigma_sqr))/sqrtf(2*M_PI*sigma_sqr);
//            sum+= d_kernel[ix+iy*ksize];
//        }
//    }

//    for(int ix = -ksize/2 ; ix< ksize/2 ; ix++)
//    {
//        for(int iy = -ksize/2 ; iy< ksize/2 ; iy++)
//        {
//            d_kernel[ix+iy*ksize] = d_kernel[ix+iy*ksize]/sum;
//        }
//    }

//    cu_GaussianConvolution<<<grid, block>>> (src,
//                                             dest,
//                                             d_kernel,
//                                             stridechar,
//                                             width,
//                                             height,
//                                             ksize);


//    delete d_kernel;
//}


